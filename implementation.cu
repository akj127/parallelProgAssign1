#include "hip/hip_runtime.h"
#include "implementation.h"
#include "stdio.h"

void printSubmissionInfo()
{
    // This will be published in the leaderboard on piazza
    // Please modify this field with something interesting
    char nick_name[] = "default-team";

    // Please fill in your information (for marking purposes only)
    char student_first_name[] = "John";
    char student_last_name[] = "Doe";
    char student_student_number[] = "00000000";

    // Printing out team information
    printf("*******************************************************************************************************\n");
    printf("Submission Information:\n");
    printf("\tnick_name: %s\n", nick_name);
    printf("\tstudent_first_name: %s\n", student_first_name);
    printf("\tstudent_last_name: %s\n", student_last_name);
    printf("\tstudent_student_number: %s\n", student_student_number);
}
#include <hip/hip_runtime.h>
#include <stdint.h>

#define BLOCK_SIZE 1024  // Number of threads per block

/**
 * Warp-level inclusive scan using shuffle instructions.
 */
__inline__ __device__
int32_t warp_scan(int32_t val) {
    // Inclusive scan within a warp using shfl_up_sync
    for (int offset = 1; offset < 32; offset <<= 1) {
        int32_t n = __shfl_up_sync(0xFFFFFFFF, val, offset);
        if ((threadIdx.x & 31) >= offset)
            val += n;
    }
    return val;
}

/**
 * Kernel for block-level inclusive scan using shared memory and warp-level primitives.
 */
__global__
void inclusive_scan_kernel(const int32_t* d_in, int32_t* d_out, int32_t* d_block_sums, size_t n) {
    __shared__ int32_t s_data[BLOCK_SIZE / 32];  // Shared memory to hold warp sums

    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t val = 0;

    if (gid < n)
        val = d_in[gid];

    // Perform warp-level inclusive scan
    val = warp_scan(val);

    // Write the sum of each warp to shared memory
    int lane = threadIdx.x % 32;
    int warp_id = threadIdx.x / 32;

    if (lane == 31)
        s_data[warp_id] = val;

    __syncthreads();

    // Let the first thread of each warp read the sums from shared memory
    int32_t warp_sum = 0;
    if (warp_id == 0 && threadIdx.x < blockDim.x / 32) {
        warp_sum = s_data[threadIdx.x];
        warp_sum = warp_scan(warp_sum);
        s_data[threadIdx.x] = warp_sum;
    }

    __syncthreads();

    // Each thread adds the sum of previous warps
    if (warp_id > 0)
        val += s_data[warp_id - 1];

    // Write the result to global memory
    if (gid < n)
        d_out[gid] = val;

    // Save the total sum of this block to d_block_sums
    if (threadIdx.x == blockDim.x - 1)
        d_block_sums[blockIdx.x] = val;
}

/**
 * Kernel to add scanned block sums to each element in the block.
 */
__global__
void add_block_sums(int32_t* d_data, const int32_t* d_block_sums, size_t n) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    if (blockIdx.x > 0 && gid < n) {
        d_data[gid] += d_block_sums[blockIdx.x - 1];
    }
}

/**
 * Recursive function to perform multi-level inclusive scan.
 */
void scan_impl(const int32_t* d_in, int32_t* d_out, size_t n) {
    int num_threads = BLOCK_SIZE;
    int num_blocks = (n + num_threads - 1) / num_threads;

    int32_t* d_block_sums = nullptr;
    hipMalloc(&d_block_sums, num_blocks * sizeof(int32_t));

    // First level scan
    inclusive_scan_kernel<<<num_blocks, num_threads>>>(d_in, d_out, d_block_sums, n);

    // Check for errors in kernel launch
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
        hipFree(d_block_sums);
        return;
    }

    // If there are multiple blocks, perform recursive scan on block sums
    if (num_blocks > 1) {
        int32_t* d_block_sums_scan = nullptr;
        hipMalloc(&d_block_sums_scan, num_blocks * sizeof(int32_t));

        // Recursively call scan_impl on block sums
        scan_impl(d_block_sums, d_block_sums_scan, num_blocks);

        // Add scanned block sums to each element
        add_block_sums<<<num_blocks, num_threads>>>(d_out, d_block_sums_scan, n);

        // Check for errors in kernel launch
        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("Kernel launch error: %s\n", hipGetErrorString(err));
            hipFree(d_block_sums_scan);
            hipFree(d_block_sums);
            return;
        }

        hipFree(d_block_sums_scan);
    }

    hipFree(d_block_sums);
}

/**
 * Implement your CUDA inclusive scan here. Feel free to add helper functions, kernels or allocate temporary memory.
 * However, you must not modify other files. CAUTION: make sure you synchronize your kernels properly and free all
 * allocated memory.
 *
 * @param d_input: input array on device
 * @param d_output: output array on device
 * @param size: number of elements in the input array
 */
void implementation(const int32_t *d_input, int32_t *d_output, size_t size) {
    // Perform inclusive scan
    scan_impl(d_input, d_output, size);

    // Ensure all kernels have finished executing
    hipDeviceSynchronize();
}
